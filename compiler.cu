#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>
#include "gpuHelpers.cuh"

hipModule_t module;
hipCtx_t ctx;

std::string loadPTX(std::string filename)
{
    std::ifstream file(filename);
    return std::string((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
}

hipFunction_t compilePTX(std::string filename, std::string kernelName)
{
    std::string ptxSource = loadPTX(filename);

    checkCudaDriver(hipInit(0));
    hipDevice_t device;
    checkCudaDriver(hipDeviceGet(&device, 0));
    checkCudaDriver(hipCtxCreate(&ctx, 0, device));

    char errorLog[8192] = {0};
    hipJitOption options[] = 
    {
        hipJitOptionErrorLogBuffer,
        hipJitOptionErrorLogBufferSizeBytes
    };
    void* optionVals[] = 
    {
        errorLog,
        reinterpret_cast<void*>(sizeof(errorLog))
    };

    checkCudaDriver(hipModuleLoadDataEx(&module, ptxSource.c_str(), 2, options, optionVals));

    hipFunction_t kernel;
    checkCudaDriver(hipModuleGetFunction(&kernel, module, kernelName.c_str()));
    return kernel;
}

void destroyContext()
{
    checkCudaDriver(hipModuleUnload(module));
    checkCudaDriver(hipCtxDestroy(ctx));
}
