#include "compiler.cuh"

std::string loadPTX(std::string filename) 
{
    std::ifstream file(filename);
    std::string content((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
    return content;
}

hipFunction_t compilePTX(std::string filename, std::string kernelName)
{
    std::string ptxSource = loadPTX(filename);
    hiprtcProgram prog;
    hiprtcCreateProgram(&prog, ptxSource.c_str(), filename.c_str(), 0, nullptr, nullptr);
    hiprtcCompileProgram(prog, 0, nullptr);

    size_t ptxSize;
    hiprtcGetCodeSize(prog, &ptxSize);
    std::vector<char> ptx(ptxSize);
    hiprtcGetCode(prog, ptx.data());

    checkCudaDriver(hipInit(0));
    hipDevice_t dev;
    checkCudaDriver(hipDeviceGet(&dev, 0));
    hipCtx_t ctx;
    checkCudaDriver(hipCtxCreate(&ctx, 0, dev));

    hipModule_t module;
    checkCudaDriver(hipModuleLoadData(&module, ptx.data()));
    hipFunction_t kernel;
    checkCudaDriver(hipModuleGetFunction(&kernel, module, kernelName.c_str()));

    checkCudaDriver(hipModuleUnload(module));
    checkCudaDriver(hipCtxDestroy(ctx));
    hiprtcDestroyProgram(&prog);

    return kernel;
}
