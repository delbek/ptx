#include "compiler.cuh"

#define PTX_PATH "cuda.ptx"

int main()
{
    hipFunction_t kernel = compilePTX(PTX_PATH, "vecAdd");

    float* d_a;
    float* d_b;
    float* d_c;
    unsigned N = 1024;
    checkCudaRuntime(hipMalloc(&d_a, sizeof(float) * N));
    checkCudaRuntime(hipMalloc(&d_b, sizeof(float) * N));
    checkCudaRuntime(hipMalloc(&d_c, sizeof(float) * N));

    void* args[] = { &d_a, &d_b, &d_c, &N };
    checkCudaDriver(hipModuleLaunchKernel(kernel,
        1, 1, 1,
        1024, 1, 1,
        0,
        0,
        args,
        0
    ));

    float* h_c = (float*)malloc(sizeof(float) * N);
    checkCudaRuntime(hipMemcpy(h_c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost));

    checkCudaRuntime(hipFree(d_a));
    checkCudaRuntime(hipFree(d_b));
    checkCudaRuntime(hipFree(d_c));
    delete[] h_c;
    
    destroyContext();
    return 0;
}
